#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>

#define CHECK_CUDA(call)                                                   \
    {                                                                      \
        hipError_t err = call;                                            \
        if (err != hipSuccess) {                                          \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__  \
                      << ": " << hipGetErrorString(err) << std::endl;    \
            exit(EXIT_FAILURE);                                            \
        }                                                                  \
    }

__global__ void sobelFilter(const unsigned char* input, unsigned char* output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int Gx[3][3] = {
        {-1, 0, 1},
        {-2, 0, 2},
        {-1, 0, 1}
    };

    int Gy[3][3] = {
        {-1, -2, -1},
        { 0,  0,  0},
        { 1,  2,  1}
    };

    if (x > 0 && y > 0 && x < width - 1 && y < height - 1) {
        int gx = 0, gy = 0;

        for (int i = -1; i <= 1; ++i)
            for (int j = -1; j <= 1; ++j) {
                int pixel = input[(y + i) * width + (x + j)];
                gx += Gx[i + 1][j + 1] * pixel;
                gy += Gy[i + 1][j + 1] * pixel;
            }

        int mag = sqrtf(gx * gx + gy * gy);
        output[y * width + x] = min(255, mag);
    } else {
        output[y * width + x] = 0;
    }
}

void readPGM(const std::string& filename, std::vector<unsigned char>& data, int& width, int& height) {
    std::ifstream file(filename, std::ios::binary);
    std::string line;
    file >> line; // P5
    file >> width >> height;
    int maxVal;
    file >> maxVal;
    file.ignore(256, '\n');
    data.resize(width * height);
    file.read(reinterpret_cast<char*>(data.data()), width * height);
}

void writePGM(const std::string& filename, const std::vector<unsigned char>& data, int width, int height) {
    std::ofstream file(filename, std::ios::binary);
    file << "P5\n" << width << " " << height << "\n255\n";
    file.write(reinterpret_cast<const char*>(data.data()), width * height);
}

int main(int argc, char** argv) {
    if (argc != 3) {
        std::cerr << "Usage: ./sobel input.pgm output.pgm\n";
        return 1;
    }

    int width, height;
    std::vector<unsigned char> input;
    readPGM(argv[1], input, width, height);

    std::vector<unsigned char> output(width * height);

    unsigned char *d_input, *d_output;
    size_t imageSize = width * height * sizeof(unsigned char);

    CHECK_CUDA(hipMalloc(&d_input, imageSize));
    CHECK_CUDA(hipMalloc(&d_output, imageSize));
    CHECK_CUDA(hipMemcpy(d_input, input.data(), imageSize, hipMemcpyHostToDevice));

    dim3 blockSize(16, 16);
    dim3 gridSize((width + 15) / 16, (height + 15) / 16);

    sobelFilter<<<gridSize, blockSize>>>(d_input, d_output, width, height);
    CHECK_CUDA(hipDeviceSynchronize());

    CHECK_CUDA(hipMemcpy(output.data(), d_output, imageSize, hipMemcpyDeviceToHost));

    writePGM(argv[2], output, width, height);

    CHECK_CUDA(hipFree(d_input));
    CHECK_CUDA(hipFree(d_output));

    std::cout << "Sobel filter applied successfully.\n";
    return 0;
}
